#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "gettime.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
     printf("Error at %s:%d\n",__FILE__,__LINE__); \
     exit(-1);}hipDeviceSynchronize();} while(0)

#define PI 3.1415926535897932384626433832795

template<typename Real>
__host__
static void _my_cuda_compute_fft(Real *data, int nx, int ny)
{
	my_time t;
	t.start();
	hipfftComplex *CompData = (hipfftComplex *)malloc(nx * ny * sizeof(hipfftComplex));
	for (int i = 0; i < nx; i++)
	{
		CompData[i].x = data[i];
		CompData[i].y = 0;
	}
	t.end();
	std::cout << "malloc CompData time: " << t.used_time() << "ms" << std::endl;
	hipfftComplex *devData;
	t.start();
	CUDA_CALL(hipMalloc((void **)&devData, dim * sizeof(hipfftComplex)));
	CUDA_CALL(hipMemcpy(devData, CompData, dim * sizeof(CompData), hipMemcpyHostToDevice));
	t.end();
	std::cout << "malloc devData time: " << t.used_time() << "ms" << std::endl;

	hipfftHandle plan;
	t.start();
	hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C, 1);
	t.end();
	std::cout << "malloc create plan time: " << t.used_time() << "ms" << std::endl;
	t.start();
	hipfftExecC2C(plan, devData, devData, HIPFFT_FORWARD);
	t.end();
	std::cout << "malloc ExecC2C time: " << t.used_time() << "ms" << std::endl;
	hipDeviceSynchronize();
	CUDA_CALL(hipMemcpy(CompData, devData, dim * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	for (int i = 0; i < dim / 2; i++)
	{
		data[2 * i] = CompData[i].x;
		data[2 * i + 1] = CompData[i].y;
	}
	data[1] = CompData[dim / 2].x;
	CUDA_CALL(hipfftDestroy(plan));
	CUDA_CALL(hipFree(devData));
	free(CompData);
}

int main()
{
	CUDA_CALL(hipSetDevice(4));
	float init[2][1];
	init[0][0] = 1;
	init[1][0] = 1;
	_my_cuda_compute_fft(init, 2, 1);
	for (int i = 0; i < 2; i++)
		printf("\ndata[%d]=%lf", i, init[i]);
	float data[80];
	for (int i = 0; i < 80; i++)
		data[i] = static_cast<float>(1);
	long long int start, end;
	start = getSystemTime();
	_my_cuda_compute_fft(data, 80);
	end = getSystemTime();
	printf("compute fft time: %lldms", end - start);
	for (int i = 0;i < 8; i++)
		printf("\ndata[%d]=%lf", i, data[i]);
	return 0;
}


