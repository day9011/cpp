/*
 * This program uses the device CURAND API to calculate what 
 * proportion of pseudo-random ints have low bit set.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x;
	hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state,
								unsigned int *result)
{
	int id = threadIdx.x + blockIdx.x;
	unsigned int x;
	hiprandState localState = state[id];
	x = hiprand(&localState);
	result[id] = x;
}

int main(int argc, char *argv[])
{
    unsigned int *devResults, *hostResults;
	hiprandState *devStates;
	hostResults = (unsigned int *)calloc(1, sizeof(unsigned int));
	CUDA_CALL(hipMalloc((void **)&devResults, sizeof(unsigned int)));
	CUDA_CALL(hipMemset(devResults, 0, sizeof(unsigned int)));
	CUDA_CALL(hipMalloc((void **)&devStates, sizeof(hiprandState)));
	setup_kernel<<<1, 1>>>(devStates);
	generate_kernel<<<1, 1>>>(devStates, devResults);
	CUDA_CALL(hipMemcpy(hostResults, devResults, sizeof(unsigned int), hipMemcpyDeviceToHost));
	printf("\nrand number is %d\n", hostResults);

    /* Cleanup */
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_mtgp_example PASSED\n");
    return EXIT_SUCCESS;
}


